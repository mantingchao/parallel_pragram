#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define thread_num 16

__device__ int mandel(float c_re, float c_im, int count)
{
    float z_re = c_re, z_im = c_im;
    int i;
    for (i = 0; i < count; ++i)
    {
        if (z_re * z_re + z_im * z_im > 4.f)
            break;

        float new_re = z_re * z_re - z_im * z_im;
        float new_im = 2.f * z_re * z_im;
        z_re = c_re + new_re;
        z_im = c_im + new_im;
    }

    return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_res, int width, int height, int maxIterations, int pitch, int pixelsPerThreads)
{
    // To avoid error caused by the floating number, use the following pseudo code
    for (int p = 0; p < pixelsPerThreads; p++)
    {
        int i = (blockIdx.x * blockDim.x + threadIdx.x) * pixelsPerThreads + p;
        int j = (blockIdx.y * blockDim.y + threadIdx.y);
        // if(p >= height) return;
        // for (int k = i; k < i + pixelsPerThreads; k++)
        // {
        // if(k >= width) continue;
        float x = lowerX + i * stepX;
        float y = lowerY + j * stepY;
        // int index = p * width + k;
        // d_res[index] = mandel(x, y, maxIterations);
        int *row = (int *)((char *)d_res + j * pitch);
        row[i] = mandel(x, y, maxIterations);
        // }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int *img, int resX, int resY, int maxIterations)
{

    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *d_res, *h_res;
    size_t pitch; // 6656
    int pixelsPerThreads = 20;
    int size = resX * resY * sizeof(int);
    hipHostAlloc((void **)&h_res, size, hipHostMallocDefault);         // host
    hipMallocPitch((void **)&d_res, &pitch, resX * sizeof(int), resY); // device
    hipMemcpy2D(d_res, pitch, h_res, sizeof(float) * resX, sizeof(float) * resX, resY, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(thread_num, thread_num);
    dim3 numBlocks(resX / (thread_num * pixelsPerThreads), resY / thread_num);
    mandelKernel<<<numBlocks, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, d_res, resX, resY, maxIterations, pitch, pixelsPerThreads);
    hipDeviceSynchronize();

    hipMemcpy2D(h_res, resX * sizeof(int), d_res, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost); // device to host
    memcpy(img, h_res, size);

    // free memory
    hipFree(d_res);
    hipHostFree(h_res);
}
